#include "hip/hip_runtime.h"
/**
 * @file    VecDouble.cu
 *
 * @author  btran
 *
 * @date    2020-05-03
 *
 * Copyright (c) organization
 *
 */

#include "VecDouble.hpp"
#include <hip/hip_runtime.h>

__global__ void kernelVecDouble(int *in, int *out, const int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < n) {
        out[tid] = in[tid] * 2;
        tid += blockDim.x * gridDim.x;
    }
}

void vecDouble(int *hIn, int *hOut, const int n)
{
    int *dIn;
    int *dOut;
    hipHostMalloc((void **)&dIn, n * sizeof(int));
    hipHostMalloc((void **)&dOut, n * sizeof(int));
    hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

    kernelVecDouble<<<1, n>>>(dIn, dOut, n);
    hipDeviceSynchronize();

    hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dIn);
    hipFree(dOut);
}
