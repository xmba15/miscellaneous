#include "hip/hip_runtime.h"
/**
 * @file    CudaUtils.cu
 *
 * @author  btran
 *
 */

#include "CudaUtils.cuh"

namespace cuda
{
namespace utils
{
namespace
{
__global__ void warmUpGPUKernel()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    ++idx;
}
}  // namespace

hipError_t warmUpGPU()
{
    warmUpGPUKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return hipGetLastError();
}
}  // namespace utils
}  // namespace cuda
