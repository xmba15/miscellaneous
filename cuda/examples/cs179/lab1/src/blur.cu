#include "hip/hip_runtime.h"
/*
 * CUDA blur
 * Kevin Yuh, 2014
 * Revised by Nailen Matschke, 2016
 * Revised by Loko Kung, 2018
 */

#include "blur.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

#include "cuda_header.cuh"

CUDA_CALLABLE void cuda_blur_kernel_convolution(uint thread_index, const float *gpu_raw_data, const float *gpu_blur_v,
                                                float *gpu_out_data, const unsigned int n_frames,
                                                const unsigned int blur_v_size)
{
    // TODO: Implement the necessary convolution function that should be
    //       completed for each thread_index. Use the CPU implementation in
    //       blur.cpp as a reference.
    if (thread_index < blur_v_size) {
        for (int j = 0; j <= thread_index; ++j) {
            gpu_out_data[thread_index] += gpu_raw_data[thread_index - j] * gpu_blur_v[j];
        }
    }

    if (thread_index >= blur_v_size && thread_index < n_frames) {
        for (int j = 0; j < blur_v_size; j++)
            gpu_out_data[thread_index] += gpu_raw_data[thread_index - j] * gpu_blur_v[j];
    }
}

__global__ void cuda_blur_kernel(const float *gpu_raw_data, const float *gpu_blur_v, float *gpu_out_data, int n_frames,
                                 int blur_v_size)
{
    // TODO: Compute the current thread index.
    uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // TODO: Update the while loop to handle all indices for this thread.
    //       Remember to advance the index as necessary.
    while (thread_index < n_frames) {
        // Do computation for this thread index
        cuda_blur_kernel_convolution(thread_index, gpu_raw_data, gpu_blur_v, gpu_out_data, n_frames, blur_v_size);
        // TODO: Update the thread index
        thread_index += blockDim.x * gridDim.x;
    }
}

float cuda_call_blur_kernel(const unsigned int blocks, const unsigned int threads_per_block, const float *raw_data,
                            const float *blur_v, float *out_data, const unsigned int n_frames,
                            const unsigned int blur_v_size)
{
    // Use the CUDA machinery for recording time
    hipEvent_t start_gpu, stop_gpu;
    float time_milli = -1;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);
    hipEventRecord(start_gpu);

    float *gpu_raw_data;
    hipMalloc((void **)&gpu_raw_data, n_frames * sizeof(float));
    hipMemcpy(gpu_raw_data, raw_data, n_frames * sizeof(float), hipMemcpyHostToDevice);

    float *gpu_blur_v;
    hipMalloc((void **)&gpu_blur_v, blur_v_size * sizeof(float));
    hipMemcpy(gpu_blur_v, blur_v, blur_v_size * sizeof(float), hipMemcpyHostToDevice);

    float *gpu_out_data;
    hipMalloc((void **)&gpu_out_data, n_frames * sizeof(float));

    cuda_blur_kernel<<<blocks, threads_per_block>>>(gpu_raw_data, gpu_blur_v, gpu_out_data, n_frames, blur_v_size);

    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    else
        fprintf(stderr, "No kernel error detected\n");

    hipMemcpy(out_data, gpu_out_data, n_frames * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpu_raw_data);
    hipFree(gpu_blur_v);
    hipFree(gpu_out_data);

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    hipEventElapsedTime(&time_milli, start_gpu, stop_gpu);
    return time_milli;
}
