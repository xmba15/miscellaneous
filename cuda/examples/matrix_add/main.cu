/**
 * @file    main.cpp
 *
 * @author  btran
 *
 * @date    2020-05-03
 *
 * Copyright (c) organization
 *
 */

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addMat(float *matA, float *matB, float *matC,
                       const uint64_t row, const uint64_t col)
{
    uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    uint64_t j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < row && j < col) {
        uint64_t index = i * col + j;
        matC[index] = matA[index] + matB[index];
    }
}

int main(int argc, char *argv[])
{
    const uint64_t row = 3000, col = 3000;

    hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
    hipEventCreate(&hostStart);
    hipEventCreate(&hostStop);
    hipEventCreate(&deviceStart);
    hipEventCreate(&deviceStop);

    float timeDifferenceOnHost, timeDifferenceOnDevice;

    float *a = new float[row * col];
    float *b = new float[row * col];
    float *c = new float[row * col];

    for (uint64_t i = 0; i < row; ++i) {
        for (uint64_t j = 0; j < col; ++j) {
            a[i * col + j] = i + j;
            b[i * col + j] = i + j;
        }
    }

    printf("Adding matrices on CPU...\n");
    hipEventRecord(hostStart, 0);
    for (uint64_t i = 0; i < row * col; ++i) {
        c[i] = a[i] + b[i];
    }
    hipEventRecord(hostStop, 0);
    hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);
    printf("Matrix addition over. Time taken on CPU: %5.5f\n",
           timeDifferenceOnHost);

    float *matA, *matB, *matC;

    printf("Adding matrices on GPU...\n");
    hipEventRecord(deviceStart, 0);
    hipMalloc((void **)&matA, row * col * sizeof(float));
    hipMalloc((void **)&matB, row * col * sizeof(float));
    hipMalloc((void **)&matC, row * col * sizeof(float));

    hipMemcpy(matA, a, row * col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(matB, b, row * col * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((row + 31) / 32, (col + 31) / 32);

    addMat<<<numBlocks, threadsPerBlock>>>(matA, matB, matC, row, col);
    hipDeviceSynchronize();

    hipMemcpy(c, matC, row * col * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(deviceStop, 0);
    hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);
    printf("Matrix addition over. Time taken on GPU: %5.5f\n",
           timeDifferenceOnDevice);

    hipFree(matA);
    hipFree(matB);
    hipFree(matC);
    hipEventDestroy(deviceStart);
    hipEventDestroy(deviceStop);
    hipEventDestroy(hostStart);
    hipEventDestroy(hostStop);

    delete[] a;
    delete[] b;
    delete[] c;
    return 0;
}
